#include "hip/hip_runtime.h"
#include<bits/stdc++.h> // header file for all c++ libraries
using namespace std;   // stdout library for printing values 
#include <iostream> 
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <list>
#include <thrust/copy.h>

__device__ volatile thrust::device_vector<float>derror;          // array to store all error values
thrust::host_vector<float>herror;          // array to store all error values


__device__ float err;
__device__ float b0 = 0;                   //initializing b0
__device__ float b1 = 0;                   //initializing b1
__device__ float alpha = 0.01;             //intializing error rate

__global__ void train(float *x, float *y){
    thrust::device_vector<float>dlist;
    /*Training Phase*/
    for (int i = 0; i < 20; i ++) {   // since there are 5 values and we want 4 epochs so run for loop for 20 times
        int idx = i % 5;              //for accessing index after every epoch
        float p = b0 + b1 * x[idx];  //calculating prediction
        err = p - y[idx];              // calculating error
        b0 = b0 - alpha * err;         // updating b0
        b1 = b1 - alpha * err * x[idx];// updating b1
        printf("B0=%f  B1=%f  error=%f \n", b0,b1,err);
        //cout<<"B0="<<b0<<" "<<"B1="<<b1<<" "<<"error="<<err<<endl;// printing values after every updation
        derror.push_back(err);
    }
    //thrust::copy(dlist.begin(),dlist.end(), derror.begin());
}

bool custom_sort(float a, float b) /* this custom sort function is defined to 
                                     sort on basis of min absolute value or error*/
{
    float  a1=abs(a-0);
    float  b1=abs(b-0);
    return a1<b1;
}




int main()
{
        
    /*Intialization Phase*/
    size_t bytes = 5*sizeof(float);

    float *x = (float*)malloc(bytes);
    float *y = (float*)malloc(bytes);

    float *d_x, *d_y;
    hipMalloc(&d_x, bytes);
    hipMalloc(&d_y, bytes);

    for (int i = 0; i<5; i++) {
        x[i] = i+1;
        y[i] = i+1;
        
    }

    hipMemcpy(d_x, x, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, bytes, hipMemcpyHostToDevice);
    /*Initializing Done*/

    train<<<1, 1>>>(x, y);
    std::sort(herror.begin(),herror.end(),custom_sort);//sorting based on error values
    thrust::copy(derror.begin(), derror.end(), herror.begin());
    cout<<"Final Values are: "<<"B0="<<b0<<" "<<"B1="<<b1<<" "<<"error="<<herror[0]<<endl;



    hipDeviceSynchronize();
    /*Testing Phase*/
    cout<<"Enter a test x value";
    float test;
    cin>>test;
    float pred=b0+b1*test;
    cout<<endl;
    cout<<"The value predicted by the model= "<<pred;


}